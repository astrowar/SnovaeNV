#include "hip/hip_runtime.h"



#include <stdio.h>
#include <vector>
#define real double
#define number float


#define NP 400

#define Q 1.29
#define me 0.511
#define eta 0.0
#define Mk  2.14     //  Kamiokande detector mass (kton).
#define Mimb 6.8     // IMB detector mass (kton).
#define Mbaksan 0.28 // Baksan detector mass (kton).
#define fk (1.0)
#define fimb (0.9055)
#define fb (1.0)
#define Cn    (1.0/(4 * 3.1416))
#define lnVk   14.56
#define lnVimb  15.73
#define delt 1.0  //2.3
#define timeK 20.0  //10.43
#define timeIMB 16.0   //5.9
#define timeEnd 40.0   



#define epsmin 1.0
#define epsmax 50.0 // 50.0
#define ddeps   0.1 // 5.0


// Noise's parameters

#define pk     1.0
#define pimb   1.0
#define pb     1.0
#define effK   1e-5
#define effIMB 1e-5

#define Integra( r, ff , x, x1 , x2 , dx )  for(x = (x1) ; x <= (x2) ; x+=0  ){ r = r + (dx) * ( ff )/6.0 ; x+=(dx)/2.0 ; r = r + 4*(dx) * ( ff )/6.0; x+=(dx)/2.0 ;r = r + (dx) * ( ff )/6.0; };
#define H  6.0

typedef struct LikelihoodParameter
{
	double alpha, T, ap, tp, tau1, tau2;
	double result;

	LikelihoodParameter(double _alpha, double _T, double _ap, double _tp, double _tau1, double _tau2);
} LikelihoodParameter;
typedef struct PressSchecter
{
	number ap, tp, tau1, tau2;
}PressSchecter;


__device__ number gaussian(number  x, number x0, number sigma)
{
	if (fabs(x - x0) < sigma * H) return (1.0 / (sqrt((number)2 * 3.1415928* sigma*sigma))) * exp(-0.5*pow((x - x0) / sigma, (number)2.0));
	return 0.0;
}



// Function type Press - Schecter
__device__  number Temp(number tu, number T, PressSchecter tmp) {


	if (tu > tmp.tp)
	{
		return  T * exp(-1.0 * min((((tu - tmp.tp) / tmp.tau1)), (number)12.0));
	}
	return  T;

}


__device__  number r(number T) {
	return 1.0;
}


__device__ number f_fermi(number E, number T)
{
	//if (T < 0.01) return 0;
	number ET = min(15.0,E / T);
	 
	return     1.0 / (exp(ET) + 1.0);
}


__device__ number kappa(number E) {

	const number a = 1.0 - Q / E;
	const number b = 1.0 - (2.0 * Q / E);
	const number c = (pow((number)Q, (number)2.0) - pow((number)me, (number)2.0)) / pow(E, (number)2.0);
	return a * sqrt(max(b + c, (number)0.0));
}


// Rate's neutrino - cooling component=

__device__  number Rcol(number E, number alpha, number T, number MMeff) {

	number fm;
	number kp;
	number saida = 0; 
	fm = f_fermi(E, T);
	if (fm <= 0.0) { return 0.0; }
	number alpha_t = alpha;
	kp = kappa(E);
	if (kp <= 0.0) return 0.0;
	saida = (1.22e-5) * pow(alpha_t, (number) 2.0) * MMeff * (pow(E, (number)4.0)) * fm * kp * pow(r(T), (number)2.0);
	 
	return saida;

}

__device__  number etabarK(number E) 
{
	 
	number c = 0.95*(1.0 - exp(-pow((E) / 9.3, 4.0)));
	return max(c,0.0);
	
}


__device__  number  noiseK(number E)
{
	return  effK * (gaussian(E, 6.0, 1.0) + 0.001); 
}

__device__  number StepK(number eps) {
	if (eps >  5.0) {
		return 1;
	}
	else { return 0.0; }
}

 

__device__  real LikelihoodK(number alpha, number T, PressSchecter tmp, real *LMax) {


	const number    tk[17] = { 0.0, 0.0, 0.107, 0.303, 0.324, 0.507, 0.686, 1.541, 1.728, 1.915, 9.219, 10.433, 12.439, 17.641, 20.257, 21.355, 23.814 }; // times of events;
	const number    Ek[17] = { 0.0, 20.0, 13.5, 7.5, 9.2, 12.8, 6.3, 35.4, 21, 19.8, 8.6, 13, 8.9, 6.5, 5.4, 4.6, 6.5 };   // energy of events;
	const number    Sigmak[17] = { 0.0, 2.9, 3.2, 2.0, 2.7, 2.9, 1.7, 8, 4.2, 3.2, 2.7, 2.6, 1.9, 1.6, 1.4, 1.3, 1.6 }; // standard deviation by events;
	const number    Bk[17] = { 1, 1.6e-5, 1.9e-3, 2.9e-2, 1.2e-2, 2.1e-3, 3.7e-2, 4.5e-5, 8.2e-5, 1.5e-5,            // detector's noise;
		1.5e-2, 1.9e-3, 1.6e-2, 3.8e-2, 2.9e-2, 2.8e-2, 3.8e-2 };
	 
	int  i ;
	number soma;


	real termo1, termo2;
	real prod;
	number eps;
	 

	number e1, e2;

	soma = 0.0;
	 
	 
	//number jddtp = 0.05;
	number time_end = timeEnd;
	termo1 = 0.0;
	time_end = min(3*timeEnd, tmp.tp + H * 1.0/tmp.tau1);
	number jddtp = (time_end - tmp.tp) / 30.0;
	//return (etabarK(eps)*(Cn*Rcol(eps + Q, alpha, Temp(0.0, T, tmp), Mk) + noiseK(epsmin)));
	
	{
		number Tj = Temp(0, T, tmp); 
		for (eps = epsmin; eps <= epsmax; eps = eps + ddeps)
		{
			number _etabarK = etabarK(eps); 
			termo1 += (_etabarK*(Cn*Rcol(eps + Q, alpha, Tj, Mk) + noiseK(eps)));
			 
		}
		termo1 = termo1 * tmp.tp;
	}

	for (number ti = tmp.tp; ti <= time_end; ti = ti + jddtp)
	{
		number Tj = Temp(ti, T, tmp);
		if (Tj > 0.01)
		{
			for (eps = epsmin; eps <= epsmax; eps = eps + ddeps)
			{
				termo1 += (etabarK(eps)*(Cn*Rcol(eps + Q, alpha, Tj, Mk) + noiseK(eps)))   * jddtp ;
			} 
			//Integra(termo1, (etabarK(eps)*(Cn*Rcol(eps + Q, alpha, Tj, Mk) + noiseK(eps))), eps, epsmin, epsmax, ddeps);
		}
	}
	termo1 = termo1 * ddeps;
	 
	//{
	//	number ti = 0;
	//	Integra(termo1, ProbNotIntegrate(alpha, ti, T, tmp), ti, 0, time_end, jddtp);
	//}

	prod = 1.0;
	for (i = 1; i <= 12; i++)
	{
		if (i == 6) continue;
		termo2 = 0.0;

		number Tj = Temp(tk[i], T, tmp);
		if (Tj > 0.01)
		{
			e1 = max(epsmin, Ek[i] - H * Sigmak[i]);
		    e2 = min(epsmax, Ek[i] + H * Sigmak[i]);
		    number dSigma = Sigmak[i] / 4.0;
			//Integra(termo2, RDet(eps) , eps, e1, e2, dSigma);
			Integra(termo2, (StepK(eps)* Cn * lnVk * gaussian(eps, Ek[i], Sigmak[i])*Cn*(Rcol(eps + Q, alpha, Tj, Mk) + 1.0*noiseK(eps))), eps, e1, e2, dSigma);
		}
		 
		prod = prod * (Bk[i] + termo2);
	}

	soma = exp(-termo1) *  prod;

	 
	if (soma > (*LMax)) *LMax = soma;
	return soma;
	
}





__device__ real Likelihood_combined(real alpha, real T, real ap, real tp, real tau1, real tau2)
{

	PressSchecter psch = { (number)ap, (number)tp, (number)tau1,(number)tau2 };
	real LMax = 0.0;
	return LikelihoodK(alpha, T, psch, &LMax);

}
 
__global__ void Likelihood(const LikelihoodParameter *inputParams,   double *results , unsigned int size)
{
	//int tid = threadIdx.x;
	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < size)
	{
		results[tid]   = Likelihood_combined(inputParams[tid].alpha, inputParams[tid].T, inputParams[tid].ap, inputParams[tid].tp, inputParams[tid].tau1, inputParams[tid].tau2);
		 
	}
	__syncthreads();

}


bool IsPowerOfTwo(unsigned int x)
{
	return (x & (x - 1)) == 0;
}
// Helper function for using CUDA to add vectors in parallel.
hipError_t LikelihoodList(LikelihoodParameter *a, double *results, const unsigned int size)
{
	hipError_t cudaStatus;
	
	static LikelihoodParameter *dev_a = 0;
	static double *dev_Lk = 0;
 
	unsigned int  size_truc = size;

	int blockSize;      // The launch configurator returned block size 
	int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 


	while (size_truc % 32 != 0) size_truc++;
	 
	int nth = size_truc / 64 + 1;

 

 
	if (dev_a == 0)
	{
		cudaStatus = hipSetDevice(0);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
			goto Error;
		}

		cudaStatus = hipMalloc((void**)&dev_a, size_truc * sizeof(LikelihoodParameter));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc  A failed! %i \n", size_truc * sizeof(LikelihoodParameter));
			goto Error;
		}
		cudaStatus = hipMalloc((void**)&dev_Lk, size_truc * sizeof(double));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMalloc B failed! %i \n", size_truc * sizeof(double));
			goto Error;
		}
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(LikelihoodParameter), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed! \n");
		goto Error;
	}

 
	       // The actual grid size needed, based on input size 
 
	hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, Likelihood, 0, size_truc);
	//  gridSize = (size_truc + blockSize - 1) / blockSize;

	//printf("gridSize = %i ,blockSize = %i \n", gridSize, blockSize);
	// Launch a kernel on the GPU with one thread for each element.
	  nth = size_truc / 64 + 1;

	//printf("BLOCK = %i ,THREADS_PER_BLOCK = %i \n", nth, 64);
	Likelihood <<<nth, 64 >>>(  dev_a  , dev_Lk , size );
	//Likelihood << <gridSize, blockSize >> >(dev_a, dev_Lk, size);

	hipDeviceSynchronize();

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "size trunc: %u\n", size_truc);
		fprintf(stderr, "Likelihood launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(results, dev_Lk, size * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	 

Error:
	 
	//cudaFree(dev_a);
	//cudaFree(dev_Lk);

	hipDeviceSynchronize();
	return cudaStatus;
}




void computeParams(std::vector<LikelihoodParameter> &params)
{
	static int enQueues = 0;
	unsigned int size = params.size();
	//printf("Queues to compute  = %i \n", size);

	std::vector<double> results(size, 0.0);	
	LikelihoodList(params.data(), results.data(), size);
	for(unsigned int k = 0;k< size ;++k)
	{
		//if (k==0) printf("result  = %g  \n" ,results[k]);
		params[k].result = results[k];
	}
	enQueues += size;

}